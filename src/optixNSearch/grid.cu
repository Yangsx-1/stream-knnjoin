#include "hip/hip_runtime.h"
/* Parially adapted from cuNSearch (https://github.com/InteractiveComputerGraphics/cuNSearch),
   especially https://github.com/InteractiveComputerGraphics/cuNSearch/blob/master/src/cuNSearchKernels.cu */

#include <sutil/vec_math.h>

#include "helper_mortonCode.h"
#include "helper_linearIndex.h"
#include "grid.h"

#include <stdio.h>

/* GPU code */
inline __host__ __device__ uint ToCellIndex_MortonMetaGrid(const GridInfo &GridInfo, int3 gridCell)
{
  //int3 temp = gridCell;

  int3 metaGridCell = make_int3(
    gridCell.x / GridInfo.meta_grid_dim,
    gridCell.y / GridInfo.meta_grid_dim,
    gridCell.z / GridInfo.meta_grid_dim);

  gridCell.x %= GridInfo.meta_grid_dim;
  gridCell.y %= GridInfo.meta_grid_dim;
  gridCell.z %= GridInfo.meta_grid_dim;
  uint metaGridIndex = CellIndicesToLinearIndex(GridInfo.MetaGridDimension, metaGridCell);

  //if (temp.x == 283 && temp.y == 10 && temp.z == 418)
  //  printf("(%d, %d, %d), (%d, %d, %d), %u, %u, %u\n", metaGridCell.x, metaGridCell.y, metaGridCell.z, gridCell.x, gridCell.y, gridCell.z, metaGridIndex, metaGridIndex * GridInfo.meta_grid_size, MortonCode3(gridCell.x, gridCell.y, gridCell.z));

  return metaGridIndex * GridInfo.meta_grid_size + MortonCode3(gridCell.x, gridCell.y, gridCell.z);
}

inline __host__ __device__
float getWidthFromIter(int iter, float cellSize) {
  // to be absolutely certain, we add 2 (not 1) to iter to accommodate points
  // at the edges of the central cell. width means there are K points within
  // the width^3 AABB, whose center is the center point of the current cell.
  // for corner points in the cell, its width^3 AABB might have less than count
  // # of points if the point distrition becomes dramatically sparse outside of
  // the current AABB. we empirically observe no issue with >1M points, but
  // with about ~100K points this could be an issue.

  return (iter * 2 + 2) * cellSize;
}

inline __host__ __device__
unsigned int getCellIdx(GridInfo gridInfo, int ix, int iy, int iz, bool morton) {
  if (morton) // z-order sort
    return ToCellIndex_MortonMetaGrid(gridInfo, make_int3(ix, iy, iz));
  else // raster order
    return (ix * gridInfo.GridDimension.y + iy) * gridInfo.GridDimension.z + iz;
}

inline __host__ __device__
bool oob(GridInfo gridInfo, int ix, int iy, int iz) { // out of boundary
  if (ix < 0 || ix >= (int)gridInfo.GridDimension.x
   || iy < 0 || iy >= (int)gridInfo.GridDimension.y
   || iz < 0 || iz >= (int)gridInfo.GridDimension.z)
    return true;
  else return false;
}

inline __host__ __device__
void addCount(unsigned int& count, unsigned int* CellParticleCounts, GridInfo gridInfo, int ix, int iy, int iz, bool morton) {
    if (oob(gridInfo, ix, iy, iz)) return;

    // TODO: weird bug using nvcc V10.0.130, Driver Version: 470.42.01, and CUDA Version: 11.4
    // (https://forums.developer.nvidia.com/t/weird-bug-involving-the-way-to-pass-parameters-to-kernels/183890)
    // that the returned result from getCellIdx is incorrect.
    // Fixed when using nvcc 11.3/.4, which, however, doesn't compile with thrust v101201. manually downgrading thrust.

    //unsigned int iCellIdx = getCellIdx(gridInfo, ix, iy, iz, morton);
    int3 cell = make_int3(ix, iy, iz);
    unsigned int iCellIdx;
    if (morton)
      iCellIdx = ToCellIndex_MortonMetaGrid(gridInfo, cell);
    else
      iCellIdx = (cell.x * gridInfo.GridDimension.y + cell.y) * gridInfo.GridDimension.z + cell.z;

    count += CellParticleCounts[iCellIdx];
    //if (ix == 87 && iy == 22 && iz == 358) printf("[%d, %d, %d]\n", ix, iy, iz, iCellIdx);
}

__host__ __device__
void calcSearchSize(int3 gridCell,
                    GridInfo gridInfo,
                    bool morton, 
                    unsigned int* CellParticleCounts,
                    float cellSize,
                    float maxWidth,
                    unsigned int knn,
                    int* cellMask
                   ) {
  // important that x/y/z are ints not units, as we check oob when they become negative.
  int x = gridCell.x;
  int y = gridCell.y;
  int z = gridCell.z;

  // TODO: weird bug using nvcc V10.0.130, Driver Version: 470.42.01, and CUDA Version: 11.4
  // (https://forums.developer.nvidia.com/t/weird-bug-involving-the-way-to-pass-parameters-to-kernels/183890)
  // that the returned result from getCellIdx is incorrect.
  // Fixed when using nvcc 11.3/.4, which, however, doesn't compile with thrust v101201. manually downgrading thrust.

  //unsigned int cellIndex = getCellIdx(gridInfo, x, y, z, morton);
  unsigned int cellIndex;
  if (morton)
    cellIndex = ToCellIndex_MortonMetaGrid(gridInfo, gridCell);
  else
    cellIndex = (gridCell.x * gridInfo.GridDimension.y + gridCell.y) * gridInfo.GridDimension.z + gridCell.z;


  //if (x == 283 && y == 10 && z == 418) printf("cell %d has %d particles. morton? %d\n", cellIndex, CellParticleCounts[cellIndex], morton);
  //assert(cellIndex <= numberOfCells);
  //if (CellParticleCounts[cellIndex] == 0) return; // should never hit this.

  int iter = 0;
  unsigned int count = 0;
  addCount(count, CellParticleCounts, gridInfo, x, y, z, morton);

  int xmin = x;
  int xmax = x;
  int ymin = y;
  int ymax = y;
  int zmin = z;
  int zmax = z;
 
  while(1) {
    // TODO: there could be corner cases here, e.g., maxWidth is very
    // small, cellSize will be 0 (same as uninitialized).
    // TODO: another optimization we can do is what if a query is so far away
    // from the search points? right now those queries will fall into the last
    // batch and searched using the search radius. how can we skip searches for
    // them altogether by doing something here? for that we need a different
    // maxWidth that encloses the search sphere.
    float width = getWidthFromIter(iter, cellSize);
 
    if (width > maxWidth) {
      cellMask[cellIndex] = iter;
      break;
    }
    else if (count >= (knn + 1)) {
      // + 1 because the count in CellParticleCounts includes the point
      // itself whereas our KNN search isn't going to return itself!
      cellMask[cellIndex] = iter;
      break;
    }
    else {
      iter++;
    }
 
    int ix, iy, iz;
 
    iz = zmin - 1;
    for (ix = xmin; ix <= xmax; ix++) {
      for (iy = ymin; iy <= ymax; iy++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }
 
    iz = zmax + 1;
    for (ix = xmin; ix <= xmax; ix++) {
      for (iy = ymin; iy <= ymax; iy++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }

    ix = xmin - 1;
    for (iy = ymin; iy <= ymax; iy++) {
      for (iz = zmin; iz <= zmax; iz++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }

    ix = xmax + 1;
    for (iy = ymin; iy <= ymax; iy++) {
      for (iz = zmin; iz <= zmax; iz++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }
 
    iy = ymin - 1;
    for (ix = xmin; ix <= xmax; ix++) {
      for (iz = zmin; iz <= zmax; iz++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }
 
    iy = ymax + 1;
    for (ix = xmin; ix <= xmax; ix++) {
      for (iz = zmin; iz <= zmax; iz++) {
        addCount(count, CellParticleCounts, gridInfo, ix, iy, iz, morton);
      }
    }
 
    xmin--;
    xmax++;
    ymin--;
    ymax++;
    zmin--;
    zmax++;
 
    addCount(count, CellParticleCounts, gridInfo, xmin, ymin, zmin, morton);
    addCount(count, CellParticleCounts, gridInfo, xmin, ymin, zmax, morton);
    addCount(count, CellParticleCounts, gridInfo, xmin, ymax, zmin, morton);
    addCount(count, CellParticleCounts, gridInfo, xmin, ymax, zmax, morton);
    addCount(count, CellParticleCounts, gridInfo, xmax, ymin, zmin, morton);
    addCount(count, CellParticleCounts, gridInfo, xmax, ymin, zmax, morton);
    addCount(count, CellParticleCounts, gridInfo, xmax, ymax, zmin, morton);
    addCount(count, CellParticleCounts, gridInfo, xmax, ymax, zmax, morton);
  }
}

__global__ void kComputeMinMax(
  const float3 *particles,
  unsigned int particleCount,
  int3 *minCell,
  int3 *maxCell
)
{
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= particleCount) return;
  const float3 particle = particles[particleIndex];

  int3 cell;
  // convert float to int since atomicMin/Max has no native float version
  // TODO: mind the float to int conversion issue
  cell.x = (int)floorf(particle.x); // floorf returns a float
  cell.y = (int)floorf(particle.y);
  cell.z = (int)floorf(particle.z);

  atomicMin(&(minCell->x), cell.x);
  atomicMin(&(minCell->y), cell.y);
  atomicMin(&(minCell->z), cell.z);

  atomicMax(&(maxCell->x), cell.x);
  atomicMax(&(maxCell->y), cell.y);
  atomicMax(&(maxCell->z), cell.z);

  //printf("%d %d %d Min: %d %d %d Max: %d %d %d \n", cell.x, cell.y, cell.z, minCell->x, minCell->y, minCell->z, maxCell->x, maxCell->y, maxCell->z);
}

__global__ void kInsertParticles_Raster(
  const GridInfo GridInfo,
  const float3 *particles,
  unsigned int *particleCellIndices,
  unsigned int *cellParticleCounts,
  unsigned int *localSortedIndices
)
{
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= GridInfo.ParticleCount) return;
  //printf("%u, %u\n", particleIndex, GridInfo.ParticleCount);

  float3 gridCellF = (particles[particleIndex] - GridInfo.GridMin) * GridInfo.GridDelta;
  int3 gridCell = make_int3(int(gridCellF.x), int(gridCellF.y), int(gridCellF.z));

  unsigned int cellIndex = (gridCell.x * GridInfo.GridDimension.y + gridCell.y) * GridInfo.GridDimension.z + gridCell.z;
  if (particleCellIndices)
    particleCellIndices[particleIndex] = cellIndex;

  //float3 query = particles[particleIndex];
  //float3 b = make_float3(-57.230999, 2.710000, 9.608000);
  //if (fabs(query.x - b.x) < 0.001 && fabs(query.y - b.y) < 0.001 && fabs(query.z - b.z) < 0.001) {
  //  printf("particle [%f, %f, %f], [%d, %d, %d] in cell %u\n", query.x, query.y, query.z, gridCell.x, gridCell.y, gridCell.z, cellIndex);
  //}

  // this stores the within-cell sorted indices of particles
  if (localSortedIndices)
    localSortedIndices[particleIndex] = atomicAdd(&cellParticleCounts[cellIndex], 1);
  else // if localSortedIndices is nullptr, we still need to increment cellParticleCounts
    atomicAdd(&cellParticleCounts[cellIndex], 1);

  //if (cellIndex == 6054598)
  //  printf("cell 6054598 has %u particles [%f, %f, %f]. Dist: %f\n", cellParticleCounts[cellIndex], query.x, query.y, query.z, sqrt((query.x - b.x) * (query.x - b.x) + (query.y - b.y) * (query.y - b.y) + (query.z - b.z) * (query.z - b.z)));

  //printf("%u, %u, (%d, %d, %d)\n", particleIndex, cellIndex, gridCell.x, gridCell.y, gridCell.z);
}

__global__ void kInsertParticles_Morton(
  const GridInfo GridInfo,
  const float3 *particles,
  unsigned int *particleCellIndices,
  unsigned int *cellParticleCounts,
  unsigned int *localSortedIndices
)
{
  unsigned int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= GridInfo.ParticleCount) return;

  float3 gridCellF = (particles[particleIndex] - GridInfo.GridMin) * GridInfo.GridDelta;
  int3 gridCell = make_int3(int(gridCellF.x), int(gridCellF.y), int(gridCellF.z));

  unsigned int cellIndex = ToCellIndex_MortonMetaGrid(GridInfo, gridCell);
  if (particleCellIndices)
    particleCellIndices[particleIndex] = cellIndex;

  //float3 query = particles[particleIndex];
  //float3 b = make_float3(21.618000, -0.005000, -13.505000);
  //if (fabs(query.x - b.x) < 0.001 && fabs(query.y - b.y) < 0.001 && fabs(query.z - b.z) < 0.001) {
  //  printf("particle [%f, %f, %f], [%d, %d, %d] in cell %u\n", query.x, query.y, query.z, gridCell.x, gridCell.y, gridCell.z, cellIndex);
  //}

  // this stores the within-cell sorted indices of particles
  if (localSortedIndices)
    localSortedIndices[particleIndex] = atomicAdd(&cellParticleCounts[cellIndex], 1);
  else // if localSortedIndices is nullptr, we still need to increment cellParticleCounts
    atomicAdd(&cellParticleCounts[cellIndex], 1);

  //printf("%u, %u, (%d, %d, %d)\n", particleIndex, cellIndex, gridCell.x, gridCell.y, gridCell.z);
}

__global__ void kCountingSortIndices(
  const GridInfo GridInfo,
  const uint* particleCellIndices,
  const uint* cellOffsets,
  const uint* localSortedIndices,
  uint* posInSortedPoints
)
{
  uint particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= GridInfo.ParticleCount) return;

  uint gridCellIndex = particleCellIndices[particleIndex];

  uint sortIndex = localSortedIndices[particleIndex] + cellOffsets[gridCellIndex];
  posInSortedPoints[particleIndex] = sortIndex;

  //printf("%u, %u, %u, %u, %u\n", particleIndex, gridCellIndex, localSortedIndices[particleIndex], cellOffsets[gridCellIndex], sortIndex);
}

__global__ void kCountingSortIndices_setRayMask(
  const GridInfo GridInfo,
  const uint* particleCellIndices,
  const uint* cellOffsets,
  const uint* localSortedIndices,
  uint* posInSortedPoints,
  int* cellMask,
  int* rayMask
)
{
  uint particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= GridInfo.ParticleCount) return;

  uint gridCellIndex = particleCellIndices[particleIndex];

  uint sortIndex = localSortedIndices[particleIndex] + cellOffsets[gridCellIndex];
  posInSortedPoints[particleIndex] = sortIndex;

  rayMask[particleIndex] = cellMask[gridCellIndex];

  //printf("%u, %u, %u, %u, %u\n", particleIndex, gridCellIndex, localSortedIndices[particleIndex], cellOffsets[gridCellIndex], sortIndex);
}

__global__ void kGenCellMask(GridInfo gridInfo,
                             bool morton, 
                             unsigned int* cellParticleCounts,
                             unsigned int* repQueries,
                             float3* particles,
                             float cellSize,
                             float maxWidth,
                             unsigned int knn,
                             int* cellMask
                            )
{
  uint particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (particleIndex >= gridInfo.ParticleCount) return;

  unsigned int qId = repQueries[particleIndex];
  float3 point = particles[qId];
  float3 gridCellF = (point - gridInfo.GridMin) * gridInfo.GridDelta;
  int3 gridCell = make_int3(int(gridCellF.x), int(gridCellF.y), int(gridCellF.z));

  calcSearchSize(gridCell,
                 gridInfo,
                 morton,
                 cellParticleCounts,
                 cellSize,
                 maxWidth,
                 knn,
                 cellMask
                );
}







/* CPU wrapper code */
void kComputeMinMax (unsigned int numOfBlocks, unsigned int threadsPerBlock, float3* points, unsigned int numPrims, int3* d_MinMax_0, int3* d_MinMax_1) {
  kComputeMinMax <<<numOfBlocks, threadsPerBlock>>> (
      points,
      numPrims,
      d_MinMax_0,
      d_MinMax_1
      );
}

void kInsertParticles(unsigned int numOfBlocks, unsigned int threadsPerBlock, GridInfo gridInfo, float3* points, unsigned int* d_ParticleCellIndices, unsigned int* d_CellParticleCounts, unsigned int* d_TempSortIndices, bool morton) {
  if (morton) {
    kInsertParticles_Morton <<<numOfBlocks, threadsPerBlock>>> (
        gridInfo,
        points,
        d_ParticleCellIndices,
        d_CellParticleCounts,
        d_TempSortIndices
        );
  } else {
    kInsertParticles_Raster <<<numOfBlocks, threadsPerBlock>>> (
        gridInfo,
        points,
        d_ParticleCellIndices,
        d_CellParticleCounts,
        d_TempSortIndices
        );
  }
}

void kCountingSortIndices(unsigned int numOfBlocks, unsigned int threadsPerBlock,
      GridInfo gridInfo,
      unsigned int* d_ParticleCellIndices,
      unsigned int* d_CellOffsets,
      unsigned int* d_LocalSortedIndices,
      unsigned int* d_posInSortedPoints
      ) {
  kCountingSortIndices <<<numOfBlocks, threadsPerBlock>>> (
      gridInfo,
      d_ParticleCellIndices,
      d_CellOffsets,
      d_LocalSortedIndices,
      d_posInSortedPoints
      );
}

void kCountingSortIndices_setRayMask(unsigned int numOfBlocks, unsigned int threadsPerBlock,
      GridInfo gridInfo,
      unsigned int* d_ParticleCellIndices,
      unsigned int* d_CellOffsets,
      unsigned int* d_LocalSortedIndices,
      unsigned int* d_posInSortedPoints,
      int* cellMask,
      int* rayMask
      ) {
  kCountingSortIndices_setRayMask <<<numOfBlocks, threadsPerBlock>>> (
      gridInfo,
      d_ParticleCellIndices,
      d_CellOffsets,
      d_LocalSortedIndices,
      d_posInSortedPoints,
      cellMask,
      rayMask
      );
}

void kCalcSearchSize(unsigned int numOfBlocks,
                     unsigned int threadsPerBlock,
                     GridInfo gridInfo,
                     bool morton, 
                     unsigned int* cellParticleCounts,
                     unsigned int* repQueries,
                     float3* particles,
                     float cellSize,
                     float maxWidth,
                     unsigned int knn,
                     int* cellMask
                    ) {
  kGenCellMask <<<numOfBlocks, threadsPerBlock>>> (
             gridInfo,
             morton,
             cellParticleCounts,
             repQueries,
             particles,
             cellSize,
             maxWidth,
             knn,
             cellMask
            );
}

float kGetWidthFromIter(int iter, float cellSize) {
  return getWidthFromIter(iter, cellSize);
}

__global__ void kTest(GridInfo gridInfo, int3 test, unsigned int* res, bool morton) {
  *res = getCellIdx(gridInfo, test.x, test.y, test.z, true);
}

void test(GridInfo gridInfo) {
  int3 test = make_int3(283, 10, 418);
  //unsigned int h_res_cpu = getCellIdx(gridInfo, test.x, test.y, test.z, true);
  //printf("%d\n", h_res_cpu);

  unsigned int* d_res;
  hipMalloc(reinterpret_cast<void**>(&d_res),
               sizeof(unsigned int) );
  
  kTest<<<1, 1>>> (gridInfo, test, d_res, true);
  unsigned int h_res;
  hipMemcpy(
              reinterpret_cast<void*>( &h_res ),
              d_res,
              sizeof( unsigned int ),
              hipMemcpyDeviceToHost
  );
  printf("%d\n", h_res);
}
